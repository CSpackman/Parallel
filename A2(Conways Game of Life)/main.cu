#include <iostream>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <assert.h>

int *init_grid(int width, int height)
{
    // Plus one to account for ghost corners/rows/cols
    int *grid = new int[(width) * (height)];
    for (unsigned i = 0; i < (width * height); ++i)
    {
        grid[i] = rand() % 2;
    }
    return grid;
}

// __global__ void calc_new_grid(int *A, int *B,int numARows, int numAColumns,int numBRows, int numBColumns,)
// {
// 	// If Corner 
//         //If Top Left
//         // If Top Right
//         //If Bottom Left
//         //If Bottom Right 
//     //If Ghost Row
//         //Top
//         //Bottom
//     //If Ghost Col
//         //Right 
//         //Left
// }

void print_grid(int *grid, int width, int height)
{
    for (int i = 0; i < width; i++)
    {
        for (int j = 0; j < height; j++)
        {
           
            if (grid[i+j])
            {
                printf("\u2588");
                // printf("1");
            }
            if (!grid[i+j])
            {
                printf("  ");
                // printf("0");
            }
            
            // printf("%d", grid[i*j]);
        }
        printf("\n");
    }
}

int main(int argc, char **argv)
{
    int GRIDWITH = 5;
    int GRIDHEIGHT = 5;
    // 8 Digit seed
    srand(92006191);

    int *grid = init_grid(GRIDWITH, GRIDHEIGHT);

    for (size_t i = 0; i < (GRIDWITH * GRIDHEIGHT); i++)
    {
        printf("%d", grid[i]);
    }
    printf("\n \n");
    print_grid(grid,GRIDWITH,GRIDHEIGHT);

    return 1;
}