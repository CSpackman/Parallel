#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <time.h>

void init_random(int *input, int n)
{
    for (int i = 0; i < n; i++)
    {
        input[i] = 1;
    }
}

void cpu_reduction_a(int *input, int n){
    for (int stride = 1; stride < n; stride *= 2) {
        for(int j = 0; j < n - stride; j += (stride * 2)){
            input[j] += input[j + stride];
        }
    }
}

void cpu_reduction_b(int *input, int n){
    int prev_stride = n;

    for (int stride = n/2; stride > 0; stride /= 2) {
        for(int j = 0; j < stride; j ++){
            input[j] += input[j + stride];
        }

        if ((prev_stride - stride) % 2 != 0 && stride != 1) {
            input[stride - 1] += input[stride];
        }
        prev_stride = stride;
    }
}

__global__ void reductionA(int *array, int *result, int n)
{
    extern __shared__ int sdata[];
    int global_index = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;

    // load elements from global memory into shared memory, if within bounds
    if (global_index < n) {
        sdata[tid] = array[global_index];
    } else {
        sdata[tid] = 0; // padding out-of-bounds elements with zero
    }
    __syncthreads();

    // perform reduction within the shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads(); // synchronize to make sure all threads complete the step
    }

    // write the result of each block's reduction to the output array
    if (tid == 0) {
        // output[blockIdx.x] = sdata[0];
        atomicAdd(result, sdata[0]);
    }
}

__global__ void reductionB(int *array)
{
    __shared__ int partialSum[];
    int global_index = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;

    // load elements from global memory into shared memory, if within bounds
    if (global_index < n) {
        partialSum[tid] = array[global_index];
    } else {
        partialSum[tid] = 0; // padding out-of-bounds elements with zero
    }
    __syncthreads();
    
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            partialSum[tid] += partialSum[tid + s];
        }
        __syncthreads();
    }
    if (tid == 0)
    {
        array[0] = partialSum[0];
    }
}

void cpu_scan(int *input, int *output, int n) {
    output[0] = 0;
    for (int i = 1; i < n; i++) {
        output[i] = input[i - 1] + output[i - 1];      
    }
}


__global__ void segmentScanKernel(int *d_out, const int *d_in, int n) {
    extern __shared__ int temp[];
    int tid = threadIdx.x;
    int offset = 1;

    temp[2 * tid] = d_in[2 * tid];
    temp[2 * tid + 1] = d_in[2 * tid + 1];

    for (int d = n >> 1; d > 0; d >>= 1) {
        __syncthreads();
        if (tid < d) {
            int ai = offset * (2 * tid + 1) - 1;
            int bi = offset * (2 * tid + 2) - 1;
            temp[bi] += temp[ai];
        }
        offset *= 2;
    }

    if (tid == 0) {
        temp[n - 1] = 0;
    }

    for (int d = 1; d < n; d *= 2) {
        offset >>= 1;
        __syncthreads();
        if (tid < d) {
            int ai = offset * (2 * tid + 1) - 1;
            int bi = offset * (2 * tid + 2) - 1;
            int t = temp[ai];
            temp[ai] = temp[bi];
            temp[bi] += t;
        }
    }
    __syncthreads();

    d_out[2 * tid] = temp[2 * tid];
    d_out[2 * tid + 1] = temp[2 * tid + 1];
}

int main()
{

    int *array, *array2;
    int n = 65536;
    array = new int[n];
    array2 = new int [n];

    const int blockSize = 256;
    const int numBlocks = (n + blockSize - 1) / blockSize;
    
    struct timespec begin, end;
	double elapsed;

    // cpu_reduction_a test
    init_random(array, n);

    clock_gettime(CLOCK_MONOTONIC, &begin);
    cpu_reduction_a(array, n);
	clock_gettime(CLOCK_MONOTONIC, &end);

	elapsed = end.tv_sec - begin.tv_sec;
	elapsed += (end.tv_nsec - begin.tv_nsec) / 1000000000.0;
	printf("CPU Reduction A Elapsed Time: %f \n", elapsed);
    std::cout << "CPU Reduction A Result: " << array[0] << std::endl;

    // cpu_reduction_b test
    init_random(array, n);
    clock_gettime(CLOCK_MONOTONIC, &begin);
    cpu_reduction_b(array, n);
	clock_gettime(CLOCK_MONOTONIC, &end);

	elapsed = end.tv_sec - begin.tv_sec;
	elapsed += (end.tv_nsec - begin.tv_nsec) / 1000000000.0;  
    printf("CPU Reduction B Elapsed Time: %f \n", elapsed);
    std::cout << "CPU Reduction B Result: " << array[0] << std::endl;

    // REDUCTION A
    int *device_array, *device_out;
    init_random(array, n);
    hipMalloc((void **)&device_array, n * sizeof(int));
    hipMalloc((void **)&device_out, numBlocks * sizeof(int));

    hipMemcpy(device_array, array, n * sizeof(int), hipMemcpyHostToDevice);
    
    clock_gettime(CLOCK_MONOTONIC, &begin);

    reductionA<<<numBlocks, blockSize, blockSize * sizeof(int)>>>(device_array, device_out, n);
    hipDeviceSynchronize();
    hipMemcpy(array, device_array, n * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(array2, device_out, numBlocks * sizeof(int), hipMemcpyDeviceToHost);

	clock_gettime(CLOCK_MONOTONIC, &end);
    elapsed = end.tv_sec - begin.tv_sec;
	elapsed += (end.tv_nsec - begin.tv_nsec) / 1000000000.0;
    printf("GPU Reduction A Elapsed Time: %f \n", elapsed);
    std::cout << "GPU Reduction A Result: " << array2[0] << std::endl;
    // std::cout << "GPU Reduction A Result: ";
    // for (int i = 0; i < n; ++i) {
    //     std::cout << array[i] << " ";
    // }
    // std::cout << std::endl;

    // REDUCTION B
    init_random(array, n);
    hipMemcpy(device_array, array, n * sizeof(int), hipMemcpyHostToDevice);
    clock_gettime(CLOCK_MONOTONIC, &begin);

    reductionB<<<1, n>>>(device_array);
    hipDeviceSynchronize();
    hipMemcpy(array, device_array, n * sizeof(int), hipMemcpyDeviceToHost);

	clock_gettime(CLOCK_MONOTONIC, &end);
    elapsed = end.tv_sec - begin.tv_sec;
	elapsed += (end.tv_nsec - begin.tv_nsec) / 1000000000.0;
    printf("GPU Reduction B Elapsed Time: %f \n", elapsed);
    std::cout << "GPU Reduction B Result: " << array[0] << std::endl;

    // cpu_scan test
    init_random(array, n);
    clock_gettime(CLOCK_MONOTONIC, &begin);
    cpu_scan(array, array2, n);
	clock_gettime(CLOCK_MONOTONIC, &end);

	elapsed = end.tv_sec - begin.tv_sec;
	elapsed += (end.tv_nsec - begin.tv_nsec) / 1000000000.0;    
    printf("CPU Scan Elapsed Time: %f \n", elapsed);
    std::cout << "CPU Scan Result: " << array2[n-1] << std::endl;

    // std::cout << "CPU Scan Result: ";
    // for (int i = 0; i < n; i++){
    //     std::cout << array2[i] << " ";
    // }
    // std::cout << std::endl;

    // gpu scan test
    int *d_in, *d_out;
    
    init_random(array, n);
    hipMalloc((void**)&d_in, n * sizeof(int));
    hipMalloc((void**)&d_out, n * sizeof(int));

    clock_gettime(CLOCK_MONOTONIC, &begin);

    hipMemcpy(d_in, array, n * sizeof(int), hipMemcpyHostToDevice);
    segmentScanKernel<<<1, n / 2, n * sizeof(int)>>>(d_out, d_in, n);
    hipDeviceSynchronize();
    hipMemcpy(array, d_out, n * sizeof(int), hipMemcpyDeviceToHost);

	clock_gettime(CLOCK_MONOTONIC, &end);
	elapsed = end.tv_sec - begin.tv_sec;
	elapsed += (end.tv_nsec - begin.tv_nsec) / 1000000000.0;    
    printf("GPU Scan Elapsed Time: %f \n", elapsed);
    std::cout << "GPU Scan Result: " << array[n-1] << std::endl;

    // std::cout << "GPU Segment scan result: ";
    // for (int i = 0; i < n; ++i) {
    //     std::cout << array[i] << " ";
    // }
    // std::cout << std::endl;
    
    hipFree(d_in);
    hipFree(d_out);
    hipFree(device_array);

    return 0;
}
